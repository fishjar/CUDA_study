#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cudastart.h"
int recursiveReduce(int *data, int const size)
{
	// terminate check
	if (size == 1) return data[0];
	// renew the stride
	int const stride = size / 2;
	if (size % 2 == 1)
	{
		for (int i = 0; i < stride; i++)
		{
			data[i] += data[i + stride];
		}
		data[0] += data[size - 1];
	}
	else
	{
		for (int i = 0; i < stride; i++)
		{
			data[i] += data[i + stride];
		}
	}
	// call
	return recursiveReduce(data, stride);
}



__global__ void reduceNeighbored(int * g_idata,int * g_odata,unsigned int n) 
{
	//set thread ID
	unsigned int tid = threadIdx.x;
	//boundary check
	if (tid >= n) return;
	//convert global data pointer to the 
	int *idata = g_idata + blockIdx.x*blockDim.x;
	//in-place reduction in global memory
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			idata[tid] += idata[tid + stride];
		}
		//synchronize within block
		__syncthreads();
	}
	//write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];

}

__global__ void reduceNeighboredLess(int * g_idata,int *g_odata,unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	// convert global data pointer to the local point of this block
	int *idata = g_idata + blockIdx.x*blockDim.x;
	if (idx > n)
		return;
	//in-place reduction in global memory
	// stride 依次 1,2,4,8...
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		// 在每个线程块有1024个线程（32个线程束）时，
		// 在第一轮迭代，前16个线程束执行计算，后16个线程束提前结束，不做计算；
		//convert tid into local array index
		// 数据还在原来的数组，但通过序号转换，计算的线程序号和数组序号不一一对应
		// 计算线程集中在前面的线程，后面的提前结束
		// 计算步骤（解决线程束发散）
		// 数据
		// 0 1 2 3 4 5 6 7
		// | | | | | | | |
		// --- --- --- ---
		// |   |   |   |
		// -----   -----
		// |       |
		// ---------
		// |
		// 线程
		// 0 1 2 3 4 5 6 7
		// | | | | | | | |
		// --- --- --- ---
		// | | | |
		// --- ---
		// | |
		// ---
		// |
		int index = 2 * stride *tid;
		if (index < blockDim.x)
		{
			idata[index] += idata[index + stride];
		}
		__syncthreads();
	}
	//write result for this block to global men
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceInterleaved(int * g_idata, int *g_odata, unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	// convert global data pointer to the local point of this block
	int *idata = g_idata + blockIdx.x*blockDim.x;
	if (idx >= n)
		return;
	//in-place reduction in global memory
	// stride 从大到小（解决数据读取不连续）
	for (int stride = blockDim.x/2; stride >0; stride >>=1)
	{
		// 计算步骤（数据和线程都集中到前面）
		// 0 1 2 3 4 5 6 7
		// | | | | | | | |
		// --- --- --- ---
		// | | | |
		// --- ---
		// | |
		// ---
		// |
		if (tid <stride)
		{
			idata[tid] += idata[tid + stride];
		}
		__syncthreads();
	}
	//write result for this block to global men
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];
}

int main(int argc,char** argv)
{
	initDevice(0);
	
	//initialization

	int size = 1 << 24;
	printf("	with array size %d  ", size);

	//execution configuration
	int blocksize = 1024;
	if (argc > 1)
	{
		blocksize = atoi(argv[1]);   //从命令行输入设置block大小
	}
	dim3 block(blocksize, 1);
	dim3 grid((size - 1) / block.x + 1, 1);
	printf("grid %d block %d \n", grid.x, block.x);

	//allocate host memory
	size_t bytes = size * sizeof(int);
	int *idata_host = (int*)malloc(bytes);
	int *odata_host = (int*)malloc(grid.x * sizeof(int));
	int * tmp = (int*)malloc(bytes);

	//initialize the array
	initialData_int(idata_host, size);

	memcpy(tmp, idata_host, bytes);
	double timeStart, timeElaps;
	int gpu_sum = 0;

	// device memory
	int * idata_dev = NULL;
	int * odata_dev = NULL;
	CHECK(hipMalloc((void**)&idata_dev, bytes));
	CHECK(hipMalloc((void**)&odata_dev, grid.x * sizeof(int)));

	//cpu reduction 对照组
	int cpu_sum = 0;
	timeStart = cpuSecond();
	//cpu_sum = recursiveReduce(tmp, size);
	for (int i = 0; i < size; i++)
		cpu_sum += tmp[i];
	timeElaps = 1000*(cpuSecond() - timeStart);

	printf("cpu sum:%d \n", cpu_sum);
	printf("cpu reduction elapsed %lf ms cpu_sum: %d\n", timeElaps, cpu_sum);


	//kernel 1 reduceNeighbored

	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceNeighbored <<<grid, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceNeighbored elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x, block.x);
    
    //kernel 2 reduceNeighboredless
	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceNeighboredLess <<<grid, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceNeighboredless elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x, block.x);

    //kernel 3 reduceInterleaved
	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceInterleaved <<<grid, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceInterleaved elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x, block.x);
    
	// free host memory

	free(idata_host);
	free(odata_host);
	CHECK(hipFree(idata_dev));
	CHECK(hipFree(odata_dev));

	//reset device
	hipDeviceReset();

	//check the results
	if (gpu_sum == cpu_sum)
	{
		printf("Test success!\n");
	}
	return EXIT_SUCCESS;
}
