#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cudastart.h"
int recursiveReduce(int *data, int const size)
{
	// terminate check
	if (size == 1) return data[0];
	// renew the stride
	int const stride = size / 2;
	if (size % 2 == 1)
	{
		for (int i = 0; i < stride; i++)
		{
			data[i] += data[i + stride];
		}
		data[0] += data[size - 1];
	}
	else
	{
		for (int i = 0; i < stride; i++)
		{
			data[i] += data[i + stride];
		}
	}
	// call
	return recursiveReduce(data, stride);
}



__global__ void reduceNeighbored(int * g_idata,int * g_odata,unsigned int n) 
{
	//set thread ID
	unsigned int tid = threadIdx.x;
	//boundary check
	if (tid >= n) return;
	//convert global data pointer to the 
	int *idata = g_idata + blockIdx.x*blockDim.x;
	//in-place reduction in global memory
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			idata[tid] += idata[tid + stride];
		}
		//synchronize within block
		__syncthreads();
	}
	//write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];

}

__global__ void reduceNeighboredLess(int * g_idata,int *g_odata,unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	// convert global data pointer to the local point of this block
	int *idata = g_idata + blockIdx.x*blockDim.x;
	if (idx > n)
		return;
	//in-place reduction in global memory
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		//convert tid into local array index
		int index = 2 * stride *tid;
		if (index < blockDim.x)
		{
			idata[index] += idata[index + stride];
		}
		__syncthreads();
	}
	//write result for this block to global men
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceInterleaved(int * g_idata, int *g_odata, unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	// convert global data pointer to the local point of this block
	int *idata = g_idata + blockIdx.x*blockDim.x;
	if (idx >= n)
		return;
	//in-place reduction in global memory
	for (int stride = blockDim.x/2; stride >0; stride >>=1)
	{
		
		if (tid <stride)
		{
			idata[tid] += idata[tid + stride];
		}
		__syncthreads();
	}
	//write result for this block to global men
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceUnroll2(int * g_idata,int * g_odata,unsigned int n)
{
	//set thread ID
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockDim.x*blockIdx.x*2+threadIdx.x;
	//boundary check
	if (idx >= n) return;
	//convert global data pointer to the
	// 将block数据的指针插入到2倍的blocksize长度
	// 即将数据分段，每段包含thread数量的2倍
	int *idata = g_idata + blockIdx.x*blockDim.x*2;
	//这一句是核心，添加来自相邻数据块的值。
	if(idx+blockDim.x<n) // 这个判断？？
	{
		// 手工执行一次加法
		// 将相邻数据块的同位置tid相加
		g_idata[idx]+=g_idata[idx+blockDim.x];

	}
	__syncthreads();
	//in-place reduction in global memory
	for (int stride = blockDim.x/2; stride>0 ; stride >>=1)
	{
		if (tid <stride)
		{
			idata[tid] += idata[tid + stride];
		}
		//synchronize within block
		__syncthreads();
	}
	//write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];

}

__global__ void reduceUnroll4(int * g_idata,int * g_odata,unsigned int n)
{
	//set thread ID
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockDim.x*blockIdx.x*4+threadIdx.x;
	//boundary check
	if (tid >= n) return;
	//convert global data pointer to the
	int *idata = g_idata + blockIdx.x*blockDim.x*4;
	if(idx+blockDim.x<n)
	{
		g_idata[idx]+=g_idata[idx+blockDim.x];
		g_idata[idx]+=g_idata[idx+blockDim.x*2];
		g_idata[idx]+=g_idata[idx+blockDim.x*3];
	}
	__syncthreads();
	//in-place reduction in global memory
	for (int stride = blockDim.x/2; stride>0 ; stride >>=1)
	{
		if (tid <stride)
		{
			idata[tid] += idata[tid + stride];
		}
		//synchronize within block
		__syncthreads();
	}
	//write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];

}

__global__ void reduceUnroll8(int * g_idata,int * g_odata,unsigned int n)
{
	//set thread ID
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockDim.x*blockIdx.x*8+threadIdx.x;
	//boundary check
	if (tid >= n) return;
	//convert global data pointer to the
	int *idata = g_idata + blockIdx.x*blockDim.x*8;
	if(idx+blockDim.x<n)
	{
		g_idata[idx]+=g_idata[idx+blockDim.x];
		g_idata[idx]+=g_idata[idx+blockDim.x*2];
		g_idata[idx]+=g_idata[idx+blockDim.x*3];
		g_idata[idx]+=g_idata[idx+blockDim.x*4];
		g_idata[idx]+=g_idata[idx+blockDim.x*5];
		g_idata[idx]+=g_idata[idx+blockDim.x*6];
		g_idata[idx]+=g_idata[idx+blockDim.x*7];

	}
	__syncthreads();
	//in-place reduction in global memory
	for (int stride = blockDim.x/2; stride>0 ; stride >>=1)
	{
		if (tid <stride)
		{
			idata[tid] += idata[tid + stride];
		}
		//synchronize within block
		__syncthreads();
	}
	//write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];

}


__global__ void reduceUnrollWarp8(int * g_idata,int * g_odata,unsigned int n)
{
	//set thread ID
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockDim.x*blockIdx.x*8+threadIdx.x;
	//boundary check
	if (tid >= n) return;
	//convert global data pointer to the
	int *idata = g_idata + blockIdx.x*blockDim.x*8;
	//unrolling 8;
	if(idx+7 * blockDim.x<n)
	{
		int a1=g_idata[idx];
		int a2=g_idata[idx+blockDim.x];
		int a3=g_idata[idx+2*blockDim.x];
		int a4=g_idata[idx+3*blockDim.x];
		int a5=g_idata[idx+4*blockDim.x];
		int a6=g_idata[idx+5*blockDim.x];
		int a7=g_idata[idx+6*blockDim.x];
		int a8=g_idata[idx+7*blockDim.x];
		g_idata[idx]=a1+a2+a3+a4+a5+a6+a7+a8;

	}
	__syncthreads();
	//in-place reduction in global memory
	for (int stride = blockDim.x/2; stride>32; stride >>=1)
	{
		if (tid <stride)
		{
			idata[tid] += idata[tid + stride];
		}
		//synchronize within block
		__syncthreads();
	}
	//write result for this block to global mem
	// 当执行到最后几次迭代时，当只需要32个或更少线程时，每次迭代后还需要进行线程束同步。
	// 为了加速，我们可以把这最后6次迭代进行展开，
	if(tid<32)
	{
		volatile int *vmem = idata;
		vmem[tid]+=vmem[tid+32];
		vmem[tid]+=vmem[tid+16];
		vmem[tid]+=vmem[tid+8];
		vmem[tid]+=vmem[tid+4];
		vmem[tid]+=vmem[tid+2];
		vmem[tid]+=vmem[tid+1];

	}

	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];

}

__global__ void reduceCompleteUnrollWarp8(int * g_idata,int * g_odata,unsigned int n)
{
	//set thread ID
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockDim.x*blockIdx.x*8+threadIdx.x;
	//boundary check
	if (tid >= n) return;
	//convert global data pointer to the
	int *idata = g_idata + blockIdx.x*blockDim.x*8;
	if(idx+7 * blockDim.x<n)
	{
		int a1=g_idata[idx];
		int a2=g_idata[idx+blockDim.x];
		int a3=g_idata[idx+2*blockDim.x];
		int a4=g_idata[idx+3*blockDim.x];
		int a5=g_idata[idx+4*blockDim.x];
		int a6=g_idata[idx+5*blockDim.x];
		int a7=g_idata[idx+6*blockDim.x];
		int a8=g_idata[idx+7*blockDim.x];
		g_idata[idx]=a1+a2+a3+a4+a5+a6+a7+a8;

	}
	__syncthreads();
	//in-place reduction in global memory
	if(blockDim.x>=1024 && tid <512)
		idata[tid]+=idata[tid+512];
	__syncthreads();
	if(blockDim.x>=512 && tid <256)
		idata[tid]+=idata[tid+256];
	__syncthreads();
	if(blockDim.x>=256 && tid <128)
		idata[tid]+=idata[tid+128];
	__syncthreads();
	if(blockDim.x>=128 && tid <64)
		idata[tid]+=idata[tid+64];
	__syncthreads();
	//write result for this block to global mem
	if(tid<32)
	{
		volatile int *vmem = idata;
		vmem[tid]+=vmem[tid+32];
		vmem[tid]+=vmem[tid+16];
		vmem[tid]+=vmem[tid+8];
		vmem[tid]+=vmem[tid+4];
		vmem[tid]+=vmem[tid+2];
		vmem[tid]+=vmem[tid+1];

	}

	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];

}

int main(int argc,char** argv)
{
	initDevice(0);
	
	//initialization

	int size = 1 << 24;
	printf("	with array size %d  ", size);

	//execution configuration
	int blocksize = 1024;
	if (argc > 1)
	{
		blocksize = atoi(argv[1]);   //从命令行输入设置block大小
	}
	dim3 block(blocksize, 1);
	dim3 grid((size - 1) / block.x + 1, 1);
	printf("grid %d block %d \n", grid.x, block.x);

	//allocate host memory
	size_t bytes = size * sizeof(int);
	int *idata_host = (int*)malloc(bytes);
	int *odata_host = (int*)malloc(grid.x * sizeof(int));
	int * tmp = (int*)malloc(bytes);

	//initialize the array
	initialData_int(idata_host, size);

	memcpy(tmp, idata_host, bytes);
	double timeStart, timeElaps;
	int gpu_sum = 0;

	// device memory
	int * idata_dev = NULL;
	int * odata_dev = NULL;
	CHECK(hipMalloc((void**)&idata_dev, bytes));
	CHECK(hipMalloc((void**)&odata_dev, grid.x * sizeof(int)));

	//cpu reduction 对照组
	int cpu_sum = 0;
	timeStart = cpuSecond();
	//cpu_sum = recursiveReduce(tmp, size);
	for (int i = 0; i < size; i++)
		cpu_sum += tmp[i];
	timeElaps = 1000*(cpuSecond() - timeStart);

	printf("cpu sum:%d \n", cpu_sum);
	printf("cpu reduction elapsed %lf ms cpu_sum: %d\n", timeElaps, cpu_sum);

    
	//kernel 1 reduceNeighbored

	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceNeighbored <<<grid, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceNeighbored elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x, block.x);
    
    //kernel 2 reduceNeighboredless
	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceNeighboredLess <<<grid, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceNeighboredless elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x, block.x);

    //kernel 3 reduceInterleaved
	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceInterleaved <<<grid, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceInterleaved elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x, block.x);
    
    //kernel 4 reduceUnroll2
	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceUnroll2 <<<grid.x/2, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x/2; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceUnroll2 elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x/2, block.x);
    
    
    //kernel 5 reduceUnroll4
	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceUnroll4 <<<grid.x/4, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x/4; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceUnroll4 elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x/4, block.x);
    
    
    //kernel 6 reduceUnroll8
	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceUnroll8 <<<grid.x/8, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x/8; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceUnroll8 elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x/8, block.x);
    
    
    //kernel 7 reduceUnrollWarp8
	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceUnrollWarp8 <<<grid.x/8, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x/8; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceUnrollWarp8 elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x/8, block.x);
    
    
    //kernel 8 reduceCompleteUnrollWarp8
	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	timeStart = cpuSecond();
	reduceCompleteUnrollWarp8 <<<grid.x/8, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x/8; i++)
		gpu_sum += odata_host[i];	
    timeElaps = 1000*(cpuSecond() - timeStart);

	printf("gpu sum:%d \n", gpu_sum);
	printf("gpu reduceCompleteUnrollWarp8 elapsed %lf ms     <<<grid %d block %d>>>\n",
		timeElaps, grid.x/8, block.x);    



	// free host memory

	free(idata_host);
	free(odata_host);
	CHECK(hipFree(idata_dev));
	CHECK(hipFree(odata_dev));

	//reset device
	hipDeviceReset();

	//check the results
	if (gpu_sum == cpu_sum)
	{
		printf("Test success!\n");
	}
	return EXIT_SUCCESS;
}
